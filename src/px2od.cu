#include "px2od.h"

px2OD::px2OD(dwContextHandle_t dwContext, hipStream_t* cudaStreamPtr)
{
    mContext = dwContext;
    SetCudaStream(cudaStreamPtr);
}

px2OD::~px2OD()
{

}

void px2OD::Init()
{
    CHECK_DW_ERROR(dwDriveNet_initDefaultParams(&mDriveNetParams));

    mDriveNetParams.maxClustersPerClass = mMaxClustersPerClass;
    mDriveNetParams.maxProposalsPerClass = mMaxProposalsPerClass;
    mDriveNetParams.networkModel = DW_DRIVENET_MODEL_FRONT;
    mDriveNetParams.batchSize = DW_DRIVENET_BATCH_SIZE_1;
    mDriveNetParams.networkPrecision = DW_PRECISION_FP32;

    CHECK_DW_ERROR(dwDriveNet_initialize(&mDriveNet, &mObjectClusteringHandles,
                                         &mDriveNetClasses,
                                         &mNumDriveNetClasses,
                                         &mDriveNetParams, mContext));

    // Initialize Object Detector from DriveNet
    CHECK_DW_ERROR(dwObjectDetector_initDefaultParams(&mDetectorParams));
    mDetectorParams.enableFuseObjects = false;
    mDetectorParams.maxNumImages = 1;

    CHECK_DW_ERROR(dwObjectDetector_initializeFromDriveNet(&mDriveNetDetector, &mDetectorParams,
                                                           mDriveNet, mContext));

    CHECK_DW_ERROR(dwObjectDetector_setCUDAStream(*mCudaStreamPtr, mDriveNetDetector));


    float32_t driveNetInputAR = 1.0f;
    dwBlobSize driveNetInputBlob;
    CHECK_DW_ERROR(dwDriveNet_getInputBlobsize(&driveNetInputBlob, mDriveNet));

    driveNetInputAR = static_cast<float32_t>(driveNetInputBlob.height) / static_cast<float32_t>(driveNetInputBlob.width);

    dwRect driveNetROI;

    driveNetROI = {0, 0, static_cast<int32_t>(CAM_IMG_WIDTH), static_cast<int32_t>(CAM_IMG_WIDTH*driveNetInputAR)};

    dwTransformation2D driveNetROITrans ={{1.0f, 0.0f, 0.0f,
                                           0.0f, 1.0f, 0.0f,
                                           0.0f, 0.0f, 1.0f}};

    CHECK_DW_ERROR(dwObjectDetector_setROI(0, &driveNetROI, &driveNetROITrans, mDriveNetDetector));

    CHECK_DW_ERROR(dwObjectDetector_getROI(&mDetectorParams.ROIs[0], &mDetectorParams.transformations[0], 0, mDriveNetDetector));

    mDetectorROI.x = mDetectorParams.ROIs[0].x;
    mDetectorROI.y = mDetectorParams.ROIs[0].y;
    mDetectorROI.width = mDetectorParams.ROIs[0].width;
    mDetectorROI.height = mDetectorParams.ROIs[0].height;

    CHECK_DW_ERROR(dwObjectDetector_bindInput(&mODInputImg, 1, mDriveNetDetector));

    for(uint32_t classIdx = 0; classIdx < mNumDriveNetClasses; ++classIdx)
    {
        mDetectorOutputObjects[classIdx].reset(new dwObjectHandle_t[MAX_OBJECT_OUTPUT_COUNT]);
        mClustererOutputObjects[classIdx].reset(new dwObjectHandle_t[MAX_OBJECT_OUTPUT_COUNT]);

        // Initialize each object handle
        for (uint32_t objIdx = 0U; objIdx < MAX_OBJECT_OUTPUT_COUNT; ++objIdx)
        {
            dwObjectData objectData{};
            dwObjectDataCamera objectDataCamera{};
            CHECK_DW_ERROR(dwObject_createCamera(&mDetectorOutputObjects[classIdx][objIdx], &objectData, &objectDataCamera));
            CHECK_DW_ERROR(dwObject_createCamera(&mClustererOutputObjects[classIdx][objIdx], &objectData, &objectDataCamera));
        }

        mDetectorOutput[classIdx].count = 0;
        mDetectorOutput[classIdx].objects = mDetectorOutputObjects[classIdx].get();
        mDetectorOutput[classIdx].maxCount = MAX_OBJECT_OUTPUT_COUNT;
        mClustererOutput[classIdx].count = 0;
        mClustererOutput[classIdx].objects = mClustererOutputObjects[classIdx].get();
        mClustererOutput[classIdx].maxCount = MAX_OBJECT_OUTPUT_COUNT;

        CHECK_DW_ERROR(dwObjectDetector_bindOutput(&mDetectorOutput[classIdx], 0, classIdx, mDriveNetDetector));
        CHECK_DW_ERROR(dwObjectClustering_bindInput(&mDetectorOutput[classIdx], mObjectClusteringHandles[classIdx]));
        CHECK_DW_ERROR(dwObjectClustering_bindOutput(&mClustererOutput[classIdx], mObjectClusteringHandles[classIdx]));
    }

    // Initialize box list
    mDnnBoxList.resize(mNumDriveNetClasses);
    mDnnLabelList.resize(mNumDriveNetClasses);
    mDnnLabelListPtr.resize(mNumDriveNetClasses);
    mDnnConfidence.resize(mNumDriveNetClasses);
    mDnnObjectID.resize(mNumDriveNetClasses);

    // Get which label name for each class id
    mClassLabels.resize(mNumDriveNetClasses);
    for(uint32_t classIdx = 0U; classIdx < mNumDriveNetClasses; ++classIdx)
    {
        const char* classLabel;
        CHECK_DW_ERROR(dwDriveNet_getClassLabel(&classLabel, classIdx, mDriveNet));
        mClassLabels[classIdx] = classLabel;

        // Reserve label and box lists
        mDnnBoxList[classIdx].reserve(mMaxClustersPerClass);
        mDnnLabelList[classIdx].reserve(mMaxClustersPerClass);
        mDnnLabelListPtr[classIdx].reserve(mMaxClustersPerClass);
        mDnnConfidence[classIdx].reserve(mMaxClustersPerClass);
        mDnnObjectID[classIdx].reserve(mMaxClustersPerClass);
    }
}

void px2OD::DetectObjects(dwImageCUDA* dwODInputImg,
                   vector<vector<dwRectf> >& outputODRectPerClass,
                   vector<const float32_t*>& outputODRectColorPerClass,
                   vector<vector<const char*> >& outputODLabelPerClass,
                   vector<vector<float32_t> >& outputODConfidencePerClass,
                   vector<vector<int> >& outputODIDPerClass)
{
    mODInputImg = dwODInputImg;
    CHECK_DW_ERROR(dwObjectDetector_processDeviceAsync(mDriveNetDetector));

    CHECK_DW_ERROR(dwObjectDetector_processHost(mDriveNetDetector));

    for (uint32_t classIdx = 0U; classIdx < mClassLabels.size(); ++classIdx)
    {
        CHECK_DW_ERROR(dwObjectClustering_process(mObjectClusteringHandles[classIdx]));

        // Get outputs of object clustering
        mDnnLabelListPtr[classIdx].clear();
        mDnnLabelList[classIdx].clear();
        mDnnBoxList[classIdx].clear();
        mDnnConfidence[classIdx].clear();
        mDnnObjectID[classIdx].clear();

        dwObjectHandleList clusters = mClustererOutput[classIdx];

        for (uint32_t objIdx = 0U; objIdx < clusters.count; ++objIdx)
        {
            dwObjectHandle_t obj = clusters.objects[objIdx];
            dwObjectDataCamera objCameraData{};
            dwObject_getDataCamera(&objCameraData, 0, obj);
            mDnnBoxList[classIdx].push_back(objCameraData.box2D);
            mDnnConfidence[classIdx].push_back(objCameraData.classConfidence);

            dwObjectData objData{};
            dwObject_getData(&objData, 0, obj);
            mDnnObjectID[classIdx].push_back(objData.id);

            string boxAnnot = mClassLabels[classIdx];
            mDnnLabelList[classIdx].push_back(boxAnnot);
            mDnnLabelListPtr[classIdx].push_back(mDnnLabelList[classIdx].back().c_str());
        }
    }

    outputODRectPerClass = mDnnBoxList;
    outputODRectColorPerClass = vector<const float*>(mOdBoxColorList, mOdBoxColorList + sizeof mOdBoxColorList/ sizeof mOdBoxColorList[0]);
    outputODLabelPerClass = mDnnLabelListPtr;
    outputODConfidencePerClass = mDnnConfidence;
    outputODIDPerClass = mDnnObjectID;
}
